#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define QX_DEF_CHAR_MAX 255

__global__ void naiveKernel(unsigned char *, float *, float *, float *, int, int, int, float);
__global__ void firstKernel(unsigned char *, float *, float *, int, int, int, float);
__global__ void secondKernel(unsigned char *, float *, float *, int, int, int, float);
__global__ void lastKernel(unsigned char *, float *, int, int);
__constant__ float *range_table_const;

void naiveRemainder(unsigned char *, float *, float *, int, int, int, float, float);

/*--------------------------------*/
/*   Naive Section                */
/*--------------------------------*/

void invokeNaiveKernel(
    unsigned char *img_h, int width, int height, int channel,
    float sigma_spatial, float sigma_range, int rows_per_block, float *buffer)
{
    unsigned char *img_d;  // image on device
    float *img_tmp_d;      // img_temp, on device
    float *map_factor_a_d; // map_factor_a, on device
    float *range_table_d;  // range table, on device

    int width_height_channel = width * height * channel;
    int width_height = width * height;
    int width_channel = width * channel;

    float *img_out_f = buffer;
    float *img_tmp_h = &img_out_f[width_height_channel];
    float *map_factor_a_h = &img_tmp_h[width_height_channel];
    float *map_factor_b = &map_factor_a_h[width_height];
    float *slice_factor_a = &map_factor_b[width_height];
    float *slice_factor_b = &slice_factor_a[width_channel];
    float *line_factor_a = &slice_factor_b[width_channel];
    float *line_factor_b = &line_factor_a[width];

    // range table for look up
    float range_table[QX_DEF_CHAR_MAX + 1];
    float inv_sigma_range = 1.0f / (sigma_range * QX_DEF_CHAR_MAX);
    for (int i = 0; i <= QX_DEF_CHAR_MAX; i++)
        range_table[i] = static_cast<float>(exp(-i * inv_sigma_range));

    // initialize on host side
    // img_tmp_h = new float[width * height * channel];
    // map_factor_a_h = new float[width * height];

    // initialize on device using cudaMalloc
    hipMalloc((void **)&range_table_d, (QX_DEF_CHAR_MAX + 1) * sizeof(float));
    if (!range_table_d)
    {
        printf("Naive Kernel: Cuda malloc fail on range_table_d");
        // delete[] img_tmp_h;
        // delete[] map_factor_a_h;
        exit(1);
    }
    hipMalloc((void **)&img_d, height * width * channel * sizeof(char));
    if (!img_d)
    {
        printf("Naive Kernel: Cuda malloc fail on img_d");
        hipFree(range_table_d);
        // delete[] img_tmp_h;
        // delete[] map_factor_a_h;
        exit(1);
    }

    hipMalloc((void **)&img_tmp_d, height * width * channel * sizeof(float));
    if (!img_tmp_d)
    {
        printf("Naive Kernel: Cuda malloc fail on img_tmp_d");
        // delete[] img_tmp_h;
        // delete[] map_factor_a_h;
        hipFree(img_d);
        hipFree(range_table_d);
        exit(1);
    }

    hipMalloc((void **)&map_factor_a_d, height * width * sizeof(float));
    if (!map_factor_a_d)
    {
        printf("Naive Kernel: Cuda malloc fail on map_factor_a_d");
        // delete[] img_tmp_h;
        // delete[] map_factor_a_h;
        hipFree(img_d);
        hipFree(img_tmp_d);
        hipFree(range_table_d);
        exit(1);
    } // finish device side initialization

    // kernel params
    int total_threads = height;
    int threads_per_block = rows_per_block;
    int num_blocks;
    if (total_threads % threads_per_block == 0)
        num_blocks = total_threads / threads_per_block;
    else
        num_blocks = total_threads / threads_per_block + 1;

    dim3 grid(num_blocks, 1, 1);
    dim3 block(threads_per_block, 1, 1);

    // copy input image to device
    hipMemcpy(img_d, img_h, height * width * channel * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(range_table_d, range_table, (QX_DEF_CHAR_MAX + 1) * sizeof(float), hipMemcpyHostToDevice);
    // invoke kernel
    naiveKernel<<<num_blocks, threads_per_block>>>(
        img_d, img_tmp_d, map_factor_a_d, range_table_d,
        width, height, channel, sigma_spatial);
    // copy back img_tmp and map_factor
    hipMemcpy(img_tmp_h, img_tmp_d, height * width * channel * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(map_factor_a_h, map_factor_a_d, height * width * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(img_d);
    hipFree(map_factor_a_d);
    // now we have img_tmp and map_factor_a on host, proceed to do the rest

    naiveRemainder(img_h, buffer, range_table, width, height, channel, sigma_spatial, sigma_range);
}

/*----- Naive Kernel -----*/

__global__ void naiveKernel(
    unsigned char *img, float *img_temp, float *map_factor_a,
    float *range_table, int width, int height, int channel,
    float sigma_spatial)
{
    int row_number = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (row_number >= height)
        return; // row index out of bound

    float *temp_x = &img_temp[row_number * width * channel];
    unsigned char *in_x = &img[row_number * width * channel];
    unsigned char *texture_x = &img[row_number * width * channel];

    float alpha = static_cast<float>(exp(-sqrt(2.0) / (sigma_spatial * width)));
    float ypr, ypg, ypb, ycr, ycg, ycb;
    float fp, fc;
    float inv_alpha_ = 1 - alpha;

    *temp_x++ = ypr = *in_x++;
    *temp_x++ = ypg = *in_x++;
    *temp_x++ = ypb = *in_x++;
    unsigned char tpr = *texture_x++;
    unsigned char tpg = *texture_x++;
    unsigned char tpb = *texture_x++;
    float *temp_factor_x = &map_factor_a[row_number * width];
    *temp_factor_x++ = fp = 1;

    for (int x = 1; x < width; x++)
    {
        unsigned char tcr = *texture_x++;
        unsigned char tcg = *texture_x++;
        unsigned char tcb = *texture_x++;
        unsigned char dr = abs(tcr - tpr);
        unsigned char dg = abs(tcg - tpg);
        unsigned char db = abs(tcb - tpb);
        int range_dist = (((dr << 1) + dg + db) >> 2);
        float weight = range_table[range_dist];
        float alpha_ = weight * alpha;
        *temp_x++ = ycr = inv_alpha_ * (*in_x++) + alpha_ * ypr;
        *temp_x++ = ycg = inv_alpha_ * (*in_x++) + alpha_ * ypg;
        *temp_x++ = ycb = inv_alpha_ * (*in_x++) + alpha_ * ypb;
        tpr = tcr;
        tpg = tcg;
        tpb = tcb;
        ypr = ycr;
        ypg = ycg;
        ypb = ycb;
        *temp_factor_x++ = fc = inv_alpha_ + alpha_ * fp;
        fp = fc;
    }
    *--temp_x;
    *temp_x = 0.5f * ((*temp_x) + (*--in_x));
    *--temp_x;
    *temp_x = 0.5f * ((*temp_x) + (*--in_x));
    *--temp_x;
    *temp_x = 0.5f * ((*temp_x) + (*--in_x));
    tpr = *--texture_x;
    tpg = *--texture_x;
    tpb = *--texture_x;
    ypr = *in_x;
    ypg = *in_x;
    ypb = *in_x;

    *--temp_factor_x;
    *temp_factor_x = 0.5f * ((*temp_factor_x) + 1);
    fp = 1;

    // from right to left
    for (int x = width - 2; x >= 0; x--)
    {
        unsigned char tcr = *--texture_x;
        unsigned char tcg = *--texture_x;
        unsigned char tcb = *--texture_x;
        unsigned char dr = abs(tcr - tpr);
        unsigned char dg = abs(tcg - tpg);
        unsigned char db = abs(tcb - tpb);
        int range_dist = (((dr << 1) + dg + db) >> 2);
        float weight = range_table[range_dist];
        float alpha_ = weight * alpha;

        ycr = inv_alpha_ * (*--in_x) + alpha_ * ypr;
        ycg = inv_alpha_ * (*--in_x) + alpha_ * ypg;
        ycb = inv_alpha_ * (*--in_x) + alpha_ * ypb;
        *--temp_x;
        *temp_x = 0.5f * ((*temp_x) + ycr);
        *--temp_x;
        *temp_x = 0.5f * ((*temp_x) + ycg);
        *--temp_x;
        *temp_x = 0.5f * ((*temp_x) + ycb);
        tpr = tcr;
        tpg = tcg;
        tpb = tcb;
        ypr = ycr;
        ypg = ycg;
        ypb = ycb;

        fc = inv_alpha_ + alpha_ * fp;
        *--temp_factor_x;
        *temp_factor_x = 0.5f * ((*temp_factor_x) + fc);
        fp = fc;
    }
}

/*----- Remaining CPU Computations -----*/
void naiveRemainder(
    unsigned char *img, float *buffer, float *range_table,
    int width, int height, int channel, float sigma_spatial, float sigma_range)
{
    int width_channel = width * channel;
    int width_height = width * height;
    int width_height_channel = width_height * channel;
    float *img_out_f = buffer;
    float *img_temp = &img_out_f[width_height_channel];
    float *map_factor_a = &img_temp[width_height_channel];
    float *map_factor_b = &map_factor_a[width_height];
    float *slice_factor_a = &map_factor_b[width_height];
    float *slice_factor_b = &slice_factor_a[width_channel];
    float *line_factor_a = &slice_factor_b[width_channel];
    float *line_factor_b = &line_factor_a[width];

    float alpha = static_cast<float>(exp(-sqrt(2.0) / (sigma_spatial * height)));
    float inv_alpha_ = 1 - alpha;
    float *ycy, *ypy, *xcy;
    unsigned char *tcy, *tpy;

    memcpy(img_out_f, img_temp, sizeof(float) * width_channel);

    float *in_factor = map_factor_a;
    float *ycf, *ypf, *xcf;
    memcpy(map_factor_b, in_factor, sizeof(float) * width);

    for (int y = 1; y < height; y++)
    {
        tpy = &img[(y - 1) * width_channel];
        tcy = &img[y * width_channel];
        xcy = &img_temp[y * width_channel];
        ypy = &img_out_f[(y - 1) * width_channel];
        ycy = &img_out_f[y * width_channel];

        xcf = &in_factor[y * width];
        ypf = &map_factor_b[(y - 1) * width];
        ycf = &map_factor_b[y * width];
        for (int x = 0; x < width; x++)
        {
            unsigned char dr = abs((*tcy++) - (*tpy++));
            unsigned char dg = abs((*tcy++) - (*tpy++));
            unsigned char db = abs((*tcy++) - (*tpy++));
            int range_dist = (((dr << 1) + dg + db) >> 2);
            float weight = range_table[range_dist];
            float alpha_ = weight * alpha;
            for (int c = 0; c < channel; c++)
                *ycy++ = inv_alpha_ * (*xcy++) + alpha_ * (*ypy++);
            *ycf++ = inv_alpha_ * (*xcf++) + alpha_ * (*ypf++);
        }
    }

    int h1 = height - 1;
    ycf = line_factor_a;
    ypf = line_factor_b;
    memcpy(ypf, &in_factor[h1 * width], sizeof(float) * width);
    for (int x = 0; x < width; x++)
        map_factor_b[h1 * width + x] = 0.5f * (map_factor_b[h1 * width + x] + ypf[x]);

    ycy = slice_factor_a;
    ypy = slice_factor_b;
    memcpy(ypy, &img_temp[h1 * width_channel], sizeof(float) * width_channel);
    int k = 0;
    for (int x = 0; x < width; x++)
    {
        for (int c = 0; c < channel; c++)
        {
            int idx = (h1 * width + x) * channel + c;
            img_out_f[idx] = 0.5f * (img_out_f[idx] + ypy[k++]) / map_factor_b[h1 * width + x];
        }
    }

    for (int y = h1 - 1; y >= 0; y--)
    {
        tpy = &img[(y + 1) * width_channel];
        tcy = &img[y * width_channel];
        xcy = &img_temp[y * width_channel];
        float *ycy_ = ycy;
        float *ypy_ = ypy;
        float *out_ = &img_out_f[y * width_channel];

        xcf = &in_factor[y * width];
        float *ycf_ = ycf;
        float *ypf_ = ypf;
        float *factor_ = &map_factor_b[y * width];
        for (int x = 0; x < width; x++)
        {
            unsigned char dr = abs((*tcy++) - (*tpy++));
            unsigned char dg = abs((*tcy++) - (*tpy++));
            unsigned char db = abs((*tcy++) - (*tpy++));
            int range_dist = (((dr << 1) + dg + db) >> 2);
            float weight = range_table[range_dist];
            float alpha_ = weight * alpha;

            float fcc = inv_alpha_ * (*xcf++) + alpha_ * (*ypf_++);
            *ycf_++ = fcc;
            *factor_ = 0.5f * (*factor_ + fcc);

            for (int c = 0; c < channel; c++)
            {
                float ycc = inv_alpha_ * (*xcy++) + alpha_ * (*ypy_++);
                *ycy_++ = ycc;
                *out_ = 0.5f * (*out_ + ycc) / (*factor_);
                *out_++;
            }
            *factor_++;
        }
        memcpy(ypy, ycy, sizeof(float) * width_channel);
        memcpy(ypf, ycf, sizeof(float) * width);
    }

    for (int i = 0; i < height * width_channel; ++i)
    {
        // printf("%.4f  ", img_out_f[i]);
        // if (i % 99 ==0) printf("\n");
        img[i] = static_cast<unsigned char>(img_out_f[i]);
    }
    // printf("\n\n");
}

/*--------------------------------*/
/*   END Naive Section            */
/*   Begin Full Refactor          */
/*--------------------------------*/

void refactorGPU(
    unsigned char *img_h, int width, int height, int channel,
    float sigma_spatial, float sigma_range, int rows_per_block)
{
    // initial steps
    unsigned char *img_d; // image on device
    float *buffer_d;      // img_out_f_d, img_tmp_d, map_factor_a_d, map_factor_b_d

    int width_height_channel = width * height * channel;
    int width_height = width * height;
    int width_channel = width * channel;

    // range table for look up
    float range_table[QX_DEF_CHAR_MAX + 1];
    float inv_sigma_range = 1.0f / (sigma_range * QX_DEF_CHAR_MAX);
    for (int i = 0; i <= QX_DEF_CHAR_MAX; i++)
        range_table[i] = static_cast<float>(exp(-i * inv_sigma_range));

    // copy range table to device
    hipMalloc((void **)&range_table_const, (QX_DEF_CHAR_MAX + 1) * sizeof(float));
    if (!range_table_const)
    {
        // delete[] range_table;
        printf("cannot allocate range table on device\n");
        exit(1);
    }
    // cudaMemcpyToSymbol(range_table_const, range_table, QX_DEF_CHAR_MAX * sizeof(float));
    // delete[] range_table;
    hipMemcpy(range_table_const, range_table, 256 * sizeof(float), hipMemcpyHostToDevice);
    // float buffer_d[img_out_f_d, img_tmp_d, map_factor_a, map_factor_b]
    int buffer_d_len = width_height_channel * 2 + width_height * 2;
    hipMalloc((void **)&buffer_d, buffer_d_len * sizeof(float));
    if (!buffer_d)
    {
        hipFree(range_table_const);
        printf("cannot allocate buffer on device\n");
        exit(1);
    }

    // copy input image to device
    hipMalloc((void **)&img_d, height * width * channel * sizeof(char));
    if (!img_d)
    {
        hipFree(range_table_const);
        hipFree(buffer_d);
        exit(1);
    }
    hipMemcpy(img_d, img_h, height * width * channel * sizeof(char), hipMemcpyHostToDevice);

    // first kernel: like first for loop in original cpu code,
    // process image left to right then right to left, each row independent
    // computes img_temp and map_factor_a
    int num_blocks = (height % rows_per_block) == 0 ? height / rows_per_block : height / rows_per_block + 1;
    dim3 grid_first(num_blocks, 1, 1);
    dim3 block_first(rows_per_block, 1, 1);
    float alpha = static_cast<float>(exp(-sqrt(2.0) / (sigma_spatial * width)));
    firstKernel<<<grid_first, block_first>>>(img_d, range_table_const, buffer_d, width, height, channel, alpha);
    // no need to copy anything back, sync and begin second kernel
    hipDeviceSynchronize();

    // second kernel: like second and third for loop in original cpu code
    // process image top to bottom, computing img_out_f and map_factor_b
    // then bottom to top, getting the final img_out_f
    alpha = static_cast<float>(exp(-sqrt(2.0) / (sigma_spatial * height)));
    num_blocks = (width % rows_per_block == 0) ? width / rows_per_block : width / rows_per_block + 1;
    dim3 grid_second(num_blocks, 1, 1);
    dim3 block_second(rows_per_block, 1, 1);
    secondKernel<<<grid_second, block_second>>>(img_d, range_table_const, buffer_d, width, height, channel, alpha);

    // last kernel: convert to unsigned char
    lastKernel<<<grid_first, block_first>>>(img_d, buffer_d, width_channel, height);
    hipMemcpy(img_h, img_d, width_height_channel * sizeof(char), hipMemcpyDeviceToHost);
}

// input: img; output: img_tmp, map_factor_a
__global__ void firstKernel(
    unsigned char *img, float *range_table, float *buffer,
    int width, int height, int channel, float alpha)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= height)
        return;

    // float alpha = static_cast<float>(exp(-sqrt(2.0) / (sigma_spatial * width)));
    float ypr, ypg, ypb, ycr, ycg, ycb;
    float fp, fc;
    float inv_alpha_ = 1 - alpha;

    // get to the parts of buffer
    float *img_tmp = &buffer[width * height * channel];
    float *map_factor_a = &img_tmp[width * height * channel];

    // get to the row
    float *temp_factor_x = &map_factor_a[row * width];
    float *temp_x = &img_tmp[row * width * channel];
    unsigned char *in_x = &img[row * width * channel];
    unsigned char *texture_x = &img[row * width * channel];

    unsigned char tpr = *texture_x++;
    unsigned char tpg = *texture_x++;
    unsigned char tpb = *texture_x++;
    *temp_factor_x++ = fp = 1;

    *temp_x++ = ypr = *in_x++;
    *temp_x++ = ypg = *in_x++;
    *temp_x++ = ypb = *in_x++;

    for (int x = 1; x < width; x++)
    {
        unsigned char tcr = *texture_x++;
        unsigned char tcg = *texture_x++;
        unsigned char tcb = *texture_x++;
        unsigned char dr = abs(tcr - tpr);
        unsigned char dg = abs(tcg - tpg);
        unsigned char db = abs(tcb - tpb);
        int range_dist = (((dr << 1) + dg + db) >> 2);
        float weight = range_table[range_dist];
        float alpha_ = weight * alpha;
        *temp_x++ = ycr = inv_alpha_ * (*in_x++) + alpha_ * ypr;
        *temp_x++ = ycg = inv_alpha_ * (*in_x++) + alpha_ * ypg;
        *temp_x++ = ycb = inv_alpha_ * (*in_x++) + alpha_ * ypb;
        tpr = tcr;
        tpg = tcg;
        tpb = tcb;
        ypr = ycr;
        ypg = ycg;
        ypb = ycb;
        *temp_factor_x++ = fc = inv_alpha_ + alpha_ * fp;
        fp = fc;
    }
    *--temp_x;
    *temp_x = 0.5f * ((*temp_x) + (*--in_x));
    *--temp_x;
    *temp_x = 0.5f * ((*temp_x) + (*--in_x));
    *--temp_x;
    *temp_x = 0.5f * ((*temp_x) + (*--in_x));
    tpr = *--texture_x;
    tpg = *--texture_x;
    tpb = *--texture_x;
    ypr = *in_x;
    ypg = *in_x;
    ypb = *in_x;

    *--temp_factor_x;
    *temp_factor_x = 0.5f * ((*temp_factor_x) + 1);
    fp = 1;

    // from right to left
    for (int x = width - 2; x >= 0; x--)
    {
        unsigned char tcr = *--texture_x;
        unsigned char tcg = *--texture_x;
        unsigned char tcb = *--texture_x;
        unsigned char dr = abs(tcr - tpr);
        unsigned char dg = abs(tcg - tpg);
        unsigned char db = abs(tcb - tpb);
        int range_dist = (((dr << 1) + dg + db) >> 2);
        float weight = range_table[range_dist];
        float alpha_ = weight * alpha;

        ycr = inv_alpha_ * (*--in_x) + alpha_ * ypr;
        ycg = inv_alpha_ * (*--in_x) + alpha_ * ypg;
        ycb = inv_alpha_ * (*--in_x) + alpha_ * ypb;
        *--temp_x;
        *temp_x = 0.5f * ((*temp_x) + ycr);
        *--temp_x;
        *temp_x = 0.5f * ((*temp_x) + ycg);
        *--temp_x;
        *temp_x = 0.5f * ((*temp_x) + ycb);
        tpr = tcr;
        tpg = tcg;
        tpb = tcb;
        ypr = ycr;
        ypg = ycg;
        ypb = ycb;

        fc = inv_alpha_ + alpha_ * fp;
        *--temp_factor_x;
        *temp_factor_x = 0.5f * ((*temp_factor_x) + fc);
        fp = fc;
    }
}

__global__ void secondKernel(
    unsigned char *img, float *range_table, float *buffer,
    int width, int height, int channel, float alpha)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= width)
        return;

    float inv_alpha_ = 1 - alpha;
    int width_channel = width * channel;
    int width_height_channel = width * height * channel;
    int width_height = width * height;

    float *ycy, *ypy, *xcy, *ycf, *ypf, *xcf;
    unsigned char *tcy, *tpy;

    float *img_out_f = buffer;
    float *img_temp = &img_out_f[width_height_channel];
    float *map_factor_a = &img_temp[width_height_channel];
    float *map_factor_b = &map_factor_a[width_height];
    float *in_factor = map_factor_a;
    tpy = &img[3 * index];
    tcy = &img[3 * index + width_channel];
    xcy = &img_temp[3 * index + width_channel];

    ypy = &img_out_f[3 * index];
    ycy = &img_out_f[3 * index + width_channel];

    xcf = &in_factor[index + width];
    ypf = &map_factor_b[index];
    ycf = &map_factor_b[index + width];

    img_out_f[3 * index] = img_temp[3 * index];
    img_out_f[3 * index + 1] = img_temp[3 * index + 1];
    img_out_f[3 * index + 2] = img_temp[3 * index + 2];
    map_factor_b[index] = map_factor_a[index];

    // top tp bottom
    for (int y = 1; y < height; y++)
    {
        unsigned char dr = abs((*tcy++) - (*tpy++));
        unsigned char dg = abs((*tcy++) - (*tpy++));
        unsigned char db = abs((*tcy++) - (*tpy++));
        int range_dist = (((dr << 1) + dg + db) >> 2);
        float weight = range_table[range_dist];
        float alpha_ = weight * alpha;
        // pointer move across column direction
        for (int c = 0; c < channel; c++)
            *ycy++ = inv_alpha_ * (*xcy++) + alpha_ * (*ypy++);
        *ycf++ = inv_alpha_ * (*xcf++) + alpha_ * (*ypf++);
        tpy = tpy - 3 + width_channel;
        tcy = tcy - 3 + width_channel;
        xcy = xcy - 3 + width_channel;

        ypy = ypy - 3 + width_channel;
        ycy = ycy - 3 + width_channel;

        xcf = xcf - 1 + width;
        ypf = ypf - 1 + width;
        ycf = ycf - 1 + width;
    }

    int h1 = height - 1, x = index;

    // in_factor = &map_factor_a[h1 * width];
    // ypf = &map_factor_b[h1 * width];
    // ypf[index] = 0.5f*(ypf[index] + in_factor[index]);
    map_factor_b[h1 * width + x] = 0.5f * (map_factor_b[h1 * width + x] + map_factor_a[h1 * width + x]);
    xcy = &img_temp[h1 * width_channel];
    ypy = &img_out_f[h1 * width_channel];

    ypy[index * 3] = 0.5f * (ypy[index * 3] + xcy[index * 3]) / ypf[index];
    ypy[index * 3 + 1] = 0.5f * (ypy[index * 3 + 1] + xcy[index * 3 + 1]) / ypf[index];
    ypy[index * 3 + 2] = 0.5f * (ypy[index * 3 + 2] + xcy[index * 3 + 2]) / ypf[index];

    tpy = &img[x * 3 + h1 * width_channel];
    tcy = tpy - width_channel;
    xcy = &img_temp[x * 3 + (h1 - 1) * width_channel];
    float at_ypf = map_factor_a[h1 * width + x];

    float at_ypy_r = img_temp[h1 * width_channel + x * 3];
    float at_ypy_g = img_temp[h1 * width_channel + x * 3 + 1];
    float at_ypy_b = img_temp[h1 * width_channel + x * 3 + 2];

    float *out_ = &img_out_f[x * 3 + (h1 - 1) * width_channel];
    xcf = &map_factor_a[(h1 - 1) * width + x];
    float *factor_ = &map_factor_b[x + (h1 - 1) * width];

    // bottom to top
    for (int y = h1 - 1; y >= 0; y--)
    {
        unsigned char dr = abs((*tcy++) - (*tpy++));
        unsigned char dg = abs((*tcy++) - (*tpy++));
        unsigned char db = abs((*tcy++) - (*tpy++));
        int range_dist = (((dr << 1) + dg + db) >> 2);
        float weight = range_table[range_dist];
        float alpha_ = weight * alpha;

        float fcc = inv_alpha_ * (*xcf) + alpha_ * (at_ypf);
        at_ypf = fcc;
        *factor_ = 0.5f * (*factor_ + fcc);

        float ycc_r = inv_alpha_ * (*xcy++) + alpha_ * at_ypy_r;
        at_ypy_r = ycc_r;

        *out_ = 0.5f * (*out_ + ycc_r) / (*factor_);
        *out_++;

        float ycc_g = inv_alpha_ * (*xcy++) + alpha_ * at_ypy_g;
        at_ypy_g = ycc_g;
        *out_ = 0.5f * (*out_ + ycc_g) / (*factor_);
        *out_++;

        float ycc_b = inv_alpha_ * (*xcy) + alpha_ * at_ypy_b;
        at_ypy_b = ycc_b;
        *out_ = 0.5f * (*out_ + ycc_b) / (*factor_);

        tcy = tcy - 3 - width_channel;
        tpy = tpy - 3 - width_channel;
        out_ = out_ - 2 - width_channel;
        xcy = xcy - 2 - width_channel;
        factor_ = factor_ - width;
        xcf = xcf - width;
    }
}

__global__ void lastKernel(unsigned char *img, float *img_out_f, int width_channel, int height)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= height)
        return;

    // int width_channel = width * channel;
    int start = row * width_channel;
    for (int i = 0; i < width_channel; i++)
    {
        img[start + i] = static_cast<unsigned char>(img_out_f[start + i]);
    }
}
